#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <math.h>
   
#define N 1000000

__global__ void counts(float *x, float *y, int *results)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(tid < N)
  {
    float result = x[tid] * x[tid] + y[tid] * y[tid];

    if(result <= 1)
      results[tid] = 1;
    else
      results[tid] = 0;
  }
}

int main(void)
{
  int T = 500; // threads per block
  int B = 2000;   // blocks per grid
  
  int *dev_results, *host_results;
  float *dev_x;
  float *dev_y;

  //host memory
  host_results = (int *) calloc(N, sizeof(float));

  //device memory
  hipMalloc((void**)&dev_x, N * sizeof(float));
  hipMalloc((void**)&dev_y, N * sizeof(float));
  hipMalloc((void**)&dev_results, N * sizeof(int));


  //random generator
  hiprandGenerator_t gen;

  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

  //generate random numbers on device
  hiprandGenerateUniform(gen, dev_x, N);
  hiprandGenerateUniform(gen, dev_y, N);

  //timmers
  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  //get results
  counts<<<B,T>>>(dev_x, dev_y, dev_results);

  //stop timers
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventRecord(stop, 0);
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //copy results to host's memory
  hipMemcpy(host_results, dev_results, N*sizeof(int), hipMemcpyDeviceToHost);

  //sum the results
  int counts = 0;
  for(int i=0;i<N;i++)
  {
    if(host_results[i])
      counts++;
    
    printf("%d ", host_results[i]);
  }
  
  float pi = 4.0 * counts / N;

  printf("Pi: %1.10f\n", pi);

  printf("Execution Time: %1.10f\n", elapsedTime);

  //cleanup
  hiprandDestroyGenerator(gen);
  hipFree(dev_x);
  hipFree(dev_y);
  hipFree(dev_results);
  free(host_results);
  return 0;
}

