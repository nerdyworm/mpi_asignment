#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define N       10 
#define THREADS 1 
#define BLOCKS  1

// size of array
__global__ void add(int *a,int *b, int *c)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid < N)
  {
    c[tid] = a[tid]+b[tid];
  }
}

int main(void)
{
  int T = 100; // threads per block
  int B = 2;   // blocks per grid

  // VS may not like defined constants in <<< >>>
  int a[N],b[N],c[N];
  int *dev_a, *dev_b, *dev_c;
  hipMalloc((void**)&dev_a,N * sizeof(int));
  hipMalloc((void**)&dev_b,N * sizeof(int));
  hipMalloc((void**)&dev_c,N * sizeof(int));

  for(int i=0;i<N;i++)
  {
    a[i] = i;
    b[i] = i*1;
  }

  hipMemcpy(dev_a, a , N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dev_c, c , N*sizeof(int),hipMemcpyHostToDevice);


  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  add<<<B,T>>>(dev_a,dev_b,dev_c);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventRecord(stop, 0);
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

  for(int i=0;i<N;i++)
  {
    printf("%d+%d=%d\n",a[i],b[i],c[i]);
  }
  
 // printf("Execution Time: %1.10f\n", elapsedTime);


  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  return 0;
}

